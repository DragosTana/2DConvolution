#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include<iostream>
#include<time.h>
#include<random>

// nvcc convolution.cu -I /usr/local/include/opencv4/ -L /usr/local/lib -lopencv_core -lopencv_imgcodecs -lopencv_highgui

#define TILE_WIDTH 32
#define w_gauss (TILE_WIDTH + KER - 1)

__constant__ float kernel[KER*KER];

void gaussian_kernel(float *kernel, int radius, float sigma){
    float sum = 0.0;
    for (int i = -radius; i <= radius; ++i){
        for (int j = -radius; j <= radius; ++j){
            kernel[(i+radius)*KER + (j+radius)] = exp(-(i*i + j*j)/(2*sigma*sigma));
            sum += kernel[(i+radius)*KER + (j+radius)];
        }
    }
    for (int i = 0; i < KER*KER; ++i){
        kernel[i] /= sum;
    }
}

void edge_detection_kernel(float *kernel){
    kernel[0] = 0; kernel[1] = -1.0; kernel[2] = 0;
    kernel[3] = -1.0; kernel[4] = 5.0; kernel[5] = -1.0;
    kernel[6] = 0; kernel[7] = -1.0; kernel[8] = 0;
}

uint64_t nanos(){
    struct timespec ts;
    clock_gettime(CLOCK_MONOTONIC, &ts);
    return (uint64_t)ts.tv_sec*1000000000 + ts.tv_nsec;
}


__global__ void smart_device_convolution(uchar *InputImageData, uchar *outputImageData, int width, int height)
{
    __shared__ uint8_t N_ds[w_gauss][w_gauss];

    int maskRadius = KER / 2;
    
    int dest = threadIdx.y * TILE_WIDTH + threadIdx.x;
    int destY = dest / w_gauss;
    int destX = dest % w_gauss;
    int srcY = blockIdx.y * TILE_WIDTH + destY - maskRadius;
    int srcX = blockIdx.x * TILE_WIDTH + destX - maskRadius;
    if (srcY >= 0 && srcY < height && srcX >= 0 && srcX < width)
        N_ds[destY][destX] = InputImageData[(srcY * width + srcX)];
    else
        N_ds[destY][destX] = 0;
        dest = threadIdx.y * TILE_WIDTH + threadIdx.x + TILE_WIDTH * TILE_WIDTH;
        destY = dest / w_gauss;
        destX = dest % w_gauss;
        srcY = blockIdx.y * TILE_WIDTH + destY - maskRadius;
        srcX = blockIdx.x * TILE_WIDTH + destX - maskRadius;
    if (destY < w_gauss)
    {
        if (srcY >= 0 && srcY < height && srcX >= 0 && srcX < width)
            N_ds[destY][destX] = InputImageData[(srcY * width + srcX)];
        else
            N_ds[destY][destX] = 0;
    }
    __syncthreads();

    uchar accum = 0;
    int y, x;
    for (y = 0; y < KER; y++)
        for (x = 0; x < KER; x++)
            accum += (uchar)N_ds[threadIdx.y + y][threadIdx.x + x] * kernel[y * KER + x];
            y = blockIdx.y * TILE_WIDTH + threadIdx.y;
            x = blockIdx.x * TILE_WIDTH + threadIdx.x;
    if (y < height && x < width)
        outputImageData[(y * width + x)] = accum;
    __syncthreads();
    
}

cv::Mat device_convolution(const cv::Mat &image, const float kernel_h[KER*KER]){
    cv::Mat output(image.rows, image.cols, CV_8UC1, cv::Scalar(0));
    uchar *d_input, *d_output;
    hipMalloc(&d_input, image.rows*image.cols*sizeof(uchar));
    hipMalloc(&d_output, image.rows*image.cols*sizeof(uchar));
    hipMemcpy(d_input, image.data, image.rows*image.cols*sizeof(uchar), hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(kernel), kernel_h, KER*KER*sizeof(float));
    dim3 dimGrid(ceil(image.cols/(float)TILE_WIDTH), ceil(image.rows/(float)TILE_WIDTH), 1);
    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);
    smart_device_convolution<<<dimGrid, dimBlock>>>(d_input, d_output, image.cols, image.rows);
    hipMemcpy(output.data, d_output, image.rows*image.cols*sizeof(uchar), hipMemcpyDeviceToHost);
    hipFree(d_input);
    hipFree(d_output);
    return output;
}
