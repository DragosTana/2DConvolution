#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void cuda_hello(){
    printf("Hello World from GPU!\n");
}

static void HandleError(hipError_t err, const char *file, int line){
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

int main() {
    cuda_hello<<<10,10>>>();
    hipDeviceSynchronize();
    return 0;
}